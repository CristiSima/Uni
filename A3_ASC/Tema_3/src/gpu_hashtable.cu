#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"
#include <hip/hip_runtime.h>

using namespace std;

/*
Allocate CUDA memory only through glbGpuAllocator
hipMalloc -> glbGpuAllocator->_cudaMalloc
hipMallocManaged -> glbGpuAllocator->_cudaMallocManaged
hipFree -> glbGpuAllocator->_cudaFree
*/



__device__ inline int hash_key(int key)
{
	#define HASH_A     6700417	
	#define HASH_B  2147483647	

	return ((long long)key*HASH_A) % HASH_B;
}

__device__ void insert_entry(table_entry *table, int table_size, table_entry entry)
{
	int key_hash= hash_key(entry.key) % table_size;

	for(int seach_index=key_hash;;seach_index++)
	{
		if(seach_index==table_size) seach_index=0;

		if(table[seach_index].key == entry.key ||
			atomicCAS((unsigned int*)&table[seach_index].key, (unsigned int)EMPTY_ENTRY, (unsigned int)entry.key) == EMPTY_ENTRY )
		{
			table[seach_index].value=entry.value;
			return;
		}
	}
}

__global__ void kerne_migrate_table(table_entry *old_table, int old_size, table_entry *new_table, int new_size)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index>=old_size)
		return;

	table_entry entry=old_table[index];

	if(entry.key == EMPTY_ENTRY)
		return;

	insert_entry(new_table, new_size, entry);
}

__global__ void kerne_insert_batch(table_entry *device_table, int table_size, table_entry *entries, int num_entries)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	index *= THREAD_CHUNK;

	for(int i=0; i<THREAD_CHUNK && index<num_entries; i++, index++)
		insert_entry(device_table, table_size, entries[index]);
}

__global__ void kernel_get_batch(table_entry *device_table, int table_size, table_entry *entries, int num_entries)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	if(index>=num_entries)
		return;

	int key = entries[index].key;
	int key_hash= hash_key(key) % table_size;

	for(int seach_index=key_hash;;seach_index++)
	{
		if(seach_index==table_size) seach_index=0;

		if(device_table[seach_index].key==key)
		{
			entries[index].value=device_table[seach_index].value;
			return;
		}
	}
}

/**
 * Function constructor GpuHashTable
 * Performs init
 * Example on using wrapper allocators _cudaMalloc and _cudaFree
 */
GpuHashTable::GpuHashTable(int size) {
	current_size=size;
	inserted_elements=0;

	glbGpuAllocator->_cudaMalloc((void**)&device_table, size * sizeof(table_entry));
	hipMemset(device_table, 0xFF, size*sizeof(table_entry));
}

/**
 * Function desctructor GpuHashTable
 */
GpuHashTable::~GpuHashTable() {
	glbGpuAllocator->_cudaFree(device_table);
}

/**
 * Function reshape
 * Performs resize of the hashtable based on load factor
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	table_entry *new_table;
	glbGpuAllocator->_cudaMalloc((void**)&new_table, numBucketsReshape * sizeof(table_entry));
	hipMemset(new_table, 0xFF, numBucketsReshape*sizeof(table_entry));

	int blocks=current_size/MAX_BLOCK_SIZE;
	blocks+=(current_size%MAX_BLOCK_SIZE > 0) ? 1 :0;
	// call kernel
	// hipError_t err=hipGetLastError();
	// printf("%d %s\n", err, hipGetErrorName(err));
	// exit(0);
	kerne_migrate_table<<<blocks, MAX_BLOCK_SIZE>>>(device_table, current_size, new_table, numBucketsReshape);
	hipDeviceSynchronize();
	
	glbGpuAllocator->_cudaFree(device_table);
	device_table=new_table;
	current_size=numBucketsReshape;
	return;
}

/**
 * Function insertBatch
 * Inserts a batch of key:value, using GPU and wrapper allocators
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	// check expand
	if(inserted_elements + numKeys >= current_size*0.9)
		reshape((inserted_elements + numKeys)* 1.75);
	
	int threads=numKeys/THREAD_CHUNK;
	threads+=(numKeys%THREAD_CHUNK > 0) ? 1 :0;

	int blocks=threads/MAX_BLOCK_SIZE;
	blocks+=(threads%MAX_BLOCK_SIZE > 0) ? 1 :0;
	
	table_entry *new_entries_d;
	glbGpuAllocator->_cudaMalloc((void**)&new_entries_d, numKeys * sizeof(table_entry));
	table_entry *new_entries_h = new table_entry[numKeys];
	for(int i=0;i<numKeys;i++)
	{
		new_entries_h[i].key=keys[i];
		new_entries_h[i].value=values[i];
	}
	hipMemcpy(new_entries_d, new_entries_h, numKeys*sizeof(new_entries_h), hipMemcpyHostToDevice);
	delete[] new_entries_h;

	kerne_insert_batch<<<blocks, MAX_BLOCK_SIZE>>>(device_table, current_size, new_entries_d, numKeys);
	hipDeviceSynchronize();

	glbGpuAllocator->_cudaFree(new_entries_d);
	inserted_elements+=numKeys;
	return true;
}

/**
 * Function getBatch
 * Gets a batch of key:value, using GPU
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	int blocks=numKeys/MAX_BLOCK_SIZE;
	blocks+=(numKeys%MAX_BLOCK_SIZE > 0) ? 1 :0;
	
	table_entry *entries;
	glbGpuAllocator->_cudaMalloc((void**)&entries, numKeys * sizeof(table_entry));

	table_entry *new_entries_h = new table_entry[numKeys];
	for(int i=0; i<numKeys; i++)
		new_entries_h[i].key=keys[i];
	hipError_t err=hipGetLastError();
	hipMemcpy(entries, new_entries_h, numKeys*sizeof(new_entries_h), hipMemcpyHostToDevice);

	kernel_get_batch<<<blocks, MAX_BLOCK_SIZE>>>(device_table, current_size, entries, numKeys);
	hipDeviceSynchronize();

	hipMemcpy(new_entries_h, entries, numKeys*sizeof(new_entries_h), hipMemcpyDeviceToHost);
	int *values=new int[numKeys];
	for(int i=0; i<numKeys; i++)
		values[i]=new_entries_h[i].value;

	int zeroes=0;
	for(int i=0; i<numKeys; i++)
		zeroes+=values[i]==0;

	delete[] new_entries_h;
	glbGpuAllocator->_cudaFree(entries);
	return values;
}
